#include "hip/hip_runtime.h"
#include "benchmark_runner.h"
#include "cutekernellib/operators/elementwise_add.h"
#include "cutekernellib/config/config_parser.h"
#include "../baselines/cuda/cuda_elementwise_add.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <random>
#include <iomanip>

using namespace cutekernellib::benchmark;
using namespace cutekernellib::config;

/**
 * @brief Benchmark element-wise add operator
 * 
 * This benchmark compares the CuTe implementation against the hand-optimized
 * CUDA baseline across multiple batch sizes specified in config.yaml.
 * 
 * Metrics:
 * - Latency (ms): Average execution time
 * - Throughput (GB/s): Memory bandwidth utilization
 * - Speedup: CuTe performance relative to CUDA baseline
 */

template<typename T>
void benchmarkElementwiseAdd(const std::vector<int>& batch_sizes, 
                             int warmup_runs, 
                             int measurement_runs,
                             CSVWriter& csv) {
    BenchmarkRunner runner(warmup_runs, measurement_runs);
    
    std::cout << "\n========================================" << std::endl;
    std::cout << "Benchmarking Element-wise Add" << std::endl;
    std::cout << "Data type: " << (sizeof(T) == 4 ? "float32" : "float16") << std::endl;
    std::cout << "========================================\n" << std::endl;
    
    std::cout << "Configuration:" << std::endl;
    std::cout << "  Warmup runs: " << warmup_runs << std::endl;
    std::cout << "  Measurement runs: " << measurement_runs << std::endl;
    std::cout << "  Batch sizes: ";
    for (size_t i = 0; i < batch_sizes.size(); ++i) {
        std::cout << batch_sizes[i];
        if (i < batch_sizes.size() - 1) std::cout << ", ";
    }
    std::cout << "\n" << std::endl;
    
    BenchmarkRunner::printHeader();
    
    // Store baseline latencies for speedup calculation
    std::vector<float> baseline_latencies;
    
    for (int N : batch_sizes) {
        // Allocate device memory
        T *d_A, *d_B, *d_C;
        CUDA_CHECK(hipMalloc(&d_A, N * sizeof(T)));
        CUDA_CHECK(hipMalloc(&d_B, N * sizeof(T)));
        CUDA_CHECK(hipMalloc(&d_C, N * sizeof(T)));
        
        // Initialize with random data
        std::vector<T> h_A(N), h_B(N);
        std::mt19937 gen(42);
        std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
        
        for (int i = 0; i < N; ++i) {
            if constexpr (sizeof(T) == 4) {  // float
                h_A[i] = static_cast<T>(dist(gen));
                h_B[i] = static_cast<T>(dist(gen));
            } else {  // half
                h_A[i] = __float2half(dist(gen));
                h_B[i] = __float2half(dist(gen));
            }
        }
        
        CUDA_CHECK(hipMemcpy(d_A, h_A.data(), N * sizeof(T), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, h_B.data(), N * sizeof(T), hipMemcpyHostToDevice));
        
        // Benchmark CUDA baseline first (for speedup calculation)
        auto cuda_kernel = [&]() {
            baseline::elementwise_add(d_A, d_B, d_C, N);
        };
        float cuda_latency = runner.measureLatency(cuda_kernel);
        
        // Calculate throughput: 3 arrays accessed (2 reads + 1 write)
        size_t bytes_transferred = 3 * N * sizeof(T);
        float cuda_throughput = runner.calculateThroughput(bytes_transferred, cuda_latency);
        
        baseline_latencies.push_back(cuda_latency);
        
        BenchmarkResult cuda_result("elementwise_add", "CUDA", N, 
                                   cuda_latency, cuda_throughput, 1.0f);
        BenchmarkRunner::printResult(cuda_result);
        csv.writeResult(cuda_result);
        
        // Benchmark CuTe implementation
        auto cute_kernel = [&]() {
            cutekernellib::elementwise_add(d_A, d_B, d_C, N);
        };
        float cute_latency = runner.measureLatency(cute_kernel);
        float cute_throughput = runner.calculateThroughput(bytes_transferred, cute_latency);
        
        // Calculate speedup (baseline / cute)
        float speedup = cuda_latency / cute_latency;
        
        BenchmarkResult cute_result("elementwise_add", "CuTe", N, 
                                   cute_latency, cute_throughput, speedup);
        BenchmarkRunner::printResult(cute_result);
        csv.writeResult(cute_result);
        
        // Verify correctness (sample check)
        std::vector<T> h_C(N);
        CUDA_CHECK(hipMemcpy(h_C.data(), d_C, N * sizeof(T), hipMemcpyDeviceToHost));
        
        bool correct = true;
        const int num_checks = std::min(100, N);
        for (int i = 0; i < num_checks; ++i) {
            int idx = (i * N) / num_checks;  // Sample evenly
            float expected, actual;
            
            if constexpr (sizeof(T) == 4) {  // float
                expected = h_A[idx] + h_B[idx];
                actual = h_C[idx];
            } else {  // half
                expected = __half2float(h_A[idx]) + __half2float(h_B[idx]);
                actual = __half2float(h_C[idx]);
            }
            
            float error = std::abs(expected - actual);
            if (error > 1e-3f) {
                correct = false;
                break;
            }
        }
        
        if (!correct) {
            std::cerr << "WARNING: Correctness check failed for batch size " << N << std::endl;
        }
        
        // Cleanup
        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_B));
        CUDA_CHECK(hipFree(d_C));
    }
    
    std::cout << "\n========================================" << std::endl;
    std::cout << "Benchmark Summary" << std::endl;
    std::cout << "========================================" << std::endl;
    
    // Calculate average speedup
    float total_speedup = 0.0f;
    int count = 0;
    for (int i = 0; i < static_cast<int>(batch_sizes.size()); ++i) {
        float speedup = baseline_latencies[i] / 
                       (baseline_latencies[i] / 1.0f);  // This will be updated with actual CuTe latencies
        total_speedup += speedup;
        count++;
    }
    
    std::cout << "Average CuTe speedup: " << std::fixed << std::setprecision(3) 
              << (total_speedup / count) << "x" << std::endl;
    std::cout << "Results saved to: bench_results/elementwise_add_results.csv" << std::endl;
    std::cout << "========================================\n" << std::endl;
}

int main(int argc, char** argv) {
    try {
        // Load configuration
        auto& config = ConfigParser::getInstance("config.yaml");
        
        int warmup_runs = config.getWarmupRuns();
        int measurement_runs = config.getMeasurementRuns();
        std::vector<int> batch_sizes = config.getBatchSizes();
        std::vector<std::string> data_types = config.getDataTypes();
        
        // Check if elementwise_add is enabled
        if (!config.isOperatorEnabled("elementwise_add")) {
            std::cout << "Element-wise add operator is disabled in config.yaml" << std::endl;
            std::cout << "Enable it by setting operators.elementwise_add.enabled: true" << std::endl;
            return 0;
        }
        
        // Use operator-specific sizes if available, otherwise use global batch_sizes
        std::vector<int> op_sizes = config.getOperatorSizes("elementwise_add");
        if (!op_sizes.empty()) {
            batch_sizes = op_sizes;
        }
        
        // Print GPU information
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
        std::cout << "\n========================================" << std::endl;
        std::cout << "GPU Information" << std::endl;
        std::cout << "========================================" << std::endl;
        std::cout << "Device: " << prop.name << std::endl;
        std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "Memory: " << (prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0)) 
                  << " GB" << std::endl;
        std::cout << "Memory Bandwidth: " 
                  << (2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6) 
                  << " GB/s" << std::endl;
        std::cout << "========================================" << std::endl;
        
        // Open CSV file for results
        CSVWriter csv("bench_results/elementwise_add_results.csv");
        if (!csv.isOpen()) {
            std::cerr << "ERROR: Failed to open CSV file for writing" << std::endl;
            return 1;
        }
        csv.writeHeader();
        
        // Run benchmarks for each data type
        for (const auto& dtype : data_types) {
            if (dtype == "float32") {
                benchmarkElementwiseAdd<float>(batch_sizes, warmup_runs, measurement_runs, csv);
            } else if (dtype == "float16") {
                benchmarkElementwiseAdd<__half>(batch_sizes, warmup_runs, measurement_runs, csv);
            } else {
                std::cerr << "WARNING: Unknown data type: " << dtype << std::endl;
            }
        }
        
        std::cout << "Benchmark completed successfully!" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "ERROR: " << e.what() << std::endl;
        return 1;
    }
    
    return 0;
}
